#include <iostream>
#include <fstream>
#include <random>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>
#include <string>
#include <cstdlib>


double G = 6.674*std::pow(10,-11);
//double G = 1;

#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__   \
                      << " - " << hipGetErrorString(err) << std::endl;    \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)


struct simulation {
  size_t nbpart;
  
  //host 
  double* hmass;
  //position
  double* hx;
  double* hy;
  double* hz;

  //velocity
  double* hvx;
  double* hvy;
  double* hvz;

  //force
  double* hfx;
  double* hfy;
  double* hfz;

  // device
  double* dmass;
      //position
  double* dx;
  double* dy;
  double* dz;

  //velocity
  double* dvx;
  double* dvy;
  double* dvz;

  //force
  double* dfx;
  double* dfy;
  double* dfz;
  

  
  simulation(size_t nb) : nbpart(nb) {
    
    //allocate host memory
    hmass = new double[nb]();
    hx = new double[nb]();
    hy = new double[nb](); 
    hz = new double[nb]();
    hvx = new double[nb](); 
    hvy = new double[nb](); 
    hvz = new double[nb]();
    hfx = new double[nb](); 
    hfy = new double[nb](); 
    hfz = new double[nb]();

    //allocate device memory
    CUDA_CHECK(hipMalloc(&dmass, nb*sizeof(double)));
    CUDA_CHECK(hipMalloc(&dx, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dy, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dz, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvx, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvy, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dvz, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfx, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfy, nb * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dfz, nb * sizeof(double)));

    //initialize memory 
    // cudaMemset(dmass, 0, nb * sizeof(double));
    // cudaMemset(dx, 0, nb * sizeof(double));
    // cudaMemset(dy, 0, nb * sizeof(double));
    // cudaMemset(dz, 0, nb * sizeof(double));
    // cudaMemset(dvx, 0, nb * sizeof(double));
    // cudaMemset(dvy, 0, nb * sizeof(double));
    // cudaMemset(dvz, 0, nb * sizeof(double));
    // cudaMemset(dfx, 0, nb * sizeof(double));
    // cudaMemset(dfy, 0, nb * sizeof(double));

  }

    //release memory
    ~simulation(){
        delete[] hmass;
        delete[] hx; 
        delete[] hy; 
        delete[] hz;
        delete[] hvx; 
        delete[] hvy; 
        delete[] hvz;
        delete[] hfx; 
        delete[] hfy; 
        delete[] hfz;

        hipFree(dmass);
        hipFree(dx);
        hipFree(dy);
        hipFree(dz);
        hipFree(dvx);
        hipFree(dvy);
        hipFree(dvz);
        hipFree(dfx);
        hipFree(dfy);
        hipFree(dfz);

    }



    void resize(size_t new_nbpart) {
        if (new_nbpart == nbpart) return;
    
            // Free old memory
        delete[] hmass;
        delete[] hx; delete[] hy; delete[] hz;
        delete[] hvx; delete[] hvy; delete[] hvz;
        delete[] hfx; delete[] hfy; delete[] hfz;

        hipFree(dmass);
        hipFree(dx); hipFree(dy); hipFree(dz);
        hipFree(dvx); hipFree(dvy); hipFree(dvz);
        hipFree(dfx); hipFree(dfy); hipFree(dfz);
    
        nbpart = new_nbpart;
        hmass = new double[nbpart]();
        hx = new double[nbpart](); 
        hy = new double[nbpart](); 
        hz = new double[nbpart]();
        hvx = new double[nbpart](); 
        hvy = new double[nbpart](); 
        hvz = new double[nbpart]();
        hfx = new double[nbpart](); 
        hfy = new double[nbpart](); 
        hfz = new double[nbpart]();


        CUDA_CHECK(hipMalloc(&dmass, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dx, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dy, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dz, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dvx, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dvy, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dvz, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dfx, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dfy, nbpart * sizeof(double)));
        CUDA_CHECK(hipMalloc(&dfz, nbpart * sizeof(double)));
    
        // cudaMemset(dmass, 0, nbpart * sizeof(double));
        // cudaMemset(dx, 0, nbpart * sizeof(double));
        // cudaMemset(dy, 0, nbpart * sizeof(double));
        // cudaMemset(dz, 0, nbpart * sizeof(double));
        // cudaMemset(dvx, 0, nbpart * sizeof(double));
        // cudaMemset(dvy, 0, nbpart * sizeof(double));
        // cudaMemset(dvz, 0, nbpart * sizeof(double));
        // cudaMemset(dfx, 0, nbpart * sizeof(double));
        // cudaMemset(dfy, 0, nbpart * sizeof(double));
        // cudaMemset(dfz, 0, nbpart * sizeof(double));
      }
    
    void host_to_device() {
        CUDA_CHECK(hipMemcpy(dmass, hmass, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dx, hx, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dy, hy, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dz, hz, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dvx, hvx, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dvy, hvy, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dvz, hvz, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dfx, hfx, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dfy, hfy, nbpart * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dfz, hfz, nbpart * sizeof(double), hipMemcpyHostToDevice));
    }

    void device_to_host() {
      CUDA_CHECK(hipMemcpy(hmass, dmass, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hx, dx, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hy, dy, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hz, dz, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hvx, dvx, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hvy, dvy, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hvz, dvz, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hfx, dfx, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hfy, dfy, nbpart * sizeof(double), hipMemcpyDeviceToHost));
      CUDA_CHECK(hipMemcpy(hfz, dfz, nbpart * sizeof(double), hipMemcpyDeviceToHost));
  }

   
};



void random_init(simulation& s) {
  std::random_device rd;  
  std::mt19937 gen(rd());
  std::uniform_real_distribution<double> dismass(0.9, 1.0);
  std::normal_distribution<double> dispos(0.0, 1.0);
  std::normal_distribution<double> disvel(0.0, 1.0);

  for (size_t i = 0; i<s.nbpart; ++i) {
    s.hmass[i] = dismass(gen);

    s.hx[i] = dispos(gen);
    s.hy[i] = dispos(gen);
    s.hz[i] = dispos(gen);
    s.hz[i] = 0.;
    
    s.hvx[i] = disvel(gen);
    s.hvy[i] = disvel(gen);
    s.hvz[i] = disvel(gen);
    s.hvz[i] = 0.;
    s.hvx[i] = s.hy[i]*1.5;
    s.hvy[i] = -s.hx[i]*1.5;
    s.hfx[i] = s.hfy[i] = s.hfz[i] = 0.0;

  }
    s.host_to_device();
}

void init_solar(simulation& s) {
  enum Planets {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE, MOON};
  s = simulation(10);

  // Masses in kg
  s.hmass[SUN] = 1.9891 * std::pow(10, 30);
  s.hmass[MERCURY] = 3.285 * std::pow(10, 23);
  s.hmass[VENUS] = 4.867 * std::pow(10, 24);
  s.hmass[EARTH] = 5.972 * std::pow(10, 24);
  s.hmass[MARS] = 6.39 * std::pow(10, 23);
  s.hmass[JUPITER] = 1.898 * std::pow(10, 27);
  s.hmass[SATURN] = 5.683 * std::pow(10, 26);
  s.hmass[URANUS] = 8.681 * std::pow(10, 25);
  s.hmass[NEPTUNE] = 1.024 * std::pow(10, 26);
  s.hmass[MOON] = 7.342 * std::pow(10, 22);

  // Positions (in meters) and velocities (in m/s)
  double AU = 1.496 * std::pow(10, 11); // Astronomical Unit

  s.hx[SUN] = 0; s.hy[SUN] = 0; s.hz[SUN] = 0;
  s.hx[MERCURY] = 0.39*AU; s.hy[MERCURY] = 0; s.hz[MERCURY] = 0;
  s.hx[VENUS] = 0.72*AU; s.hy[VENUS] = 0; s.hz[VENUS] = 0;
  s.hx[EARTH] = 1.0*AU; s.hy[EARTH] = 0; s.hz[EARTH] = 0;
  s.hx[MARS] = 1.52*AU; s.hy[MARS] = 0; s.hz[MARS] = 0;
  s.hx[JUPITER] = 5.20*AU; s.hy[JUPITER] = 0; s.hz[JUPITER] = 0;
  s.hx[SATURN] = 9.58*AU; s.hy[SATURN] = 0; s.hz[SATURN] = 0;
  s.hx[URANUS] = 19.22*AU; s.hy[URANUS] = 0; s.hz[URANUS] = 0;
  s.hx[NEPTUNE] = 30.05*AU; s.hy[NEPTUNE] = 0; s.hz[NEPTUNE] = 0;
  s.hx[MOON] = 1.0*AU + 3.844*std::pow(10, 8); s.hy[MOON] = 0; s.hz[MOON] = 0;

  s.hvx[SUN] = 0; s.hvy[SUN] = 0; s.hvz[SUN] = 0;
  s.hvx[MERCURY] = 0; s.hvy[MERCURY] = 47870; s.hvz[MERCURY] = 0;
  s.hvx[VENUS] = 0; s.hvy[VENUS] = 35020; s.hvz[VENUS] = 0;
  s.hvx[EARTH] = 0; s.hvy[EARTH] = 29780; s.hvz[EARTH] = 0;
  s.hvx[MARS] = 0; s.hvy[MARS] = 24130; s.hvz[MARS] = 0;
  s.hvx[JUPITER] = 0; s.hvy[JUPITER] = 13070; s.hvz[JUPITER] = 0;
  s.hvx[SATURN] = 0; s.hvy[SATURN] = 9680; s.hvz[SATURN] = 0;
  s.hvx[URANUS] = 0; s.hvy[URANUS] = 6800; s.hvz[URANUS] = 0;
  s.hvx[NEPTUNE] = 0; s.hvy[NEPTUNE] = 5430; s.hvz[NEPTUNE] = 0;
  s.hvx[MOON] = 0; s.hvy[MOON] = 29780 + 1022; s.hvz[MOON] = 0;

  for (int i = 0; i < 10; i++) {
    s.hfx[i] = 0;
    s.hfy[i] = 0;
    s.hfz[i] = 0;
    }   

    s.host_to_device();
}

//cuda kernel 
__global__ void update_particles_kernel(double* dx, double* dy, double* dz,
    double* dvx, double* dvy, double* dvz,
    double* dfx, double* dfy, double* dfz,
    double* dmass, size_t nbpart, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nbpart) return;

    dvx[i] += dfx[i] / dmass[i] * dt;
    dvy[i] += dfy[i] / dmass[i] * dt;
    dvz[i] += dfz[i] / dmass[i] * dt;

    dx[i] += dvx[i] * dt;
    dy[i] += dvy[i] * dt;
    dz[i] += dvz[i] * dt;
}

__global__ void reset_forces_kernel(double* dfx, double* dfy, double* dfz, size_t nbpart) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  dfx[i] = 0.;
  dfy[i] = 0.;
  dfz[i] = 0.;
}

__global__ void compute_forces_kernel(double* dmass, double* dx, double* dy, double* dz, 
    double* dfx, double* dfy, double* dfz, 
    size_t nbpart, double G) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nbpart) return;

    double softening = 0.1;
    double my_x = dx[i];
    double my_y = dy[i];
    double my_z = dz[i];
    double my_fx = 0.0;
    double my_fy = 0.0;
    double my_fz = 0.0;

    for (int j = 0; j < nbpart; j++) {
        if (i == j) continue;

        double delta_x = dx[j] - my_x;
        double delta_y = dy[j] - my_y;
        double delta_z = dz[j] - my_z;

        double dist_sq = delta_x*delta_x + delta_y*delta_y + delta_z*delta_z + softening;
        double inv_dist = rsqrt(dist_sq);
        double inv_dist3 = inv_dist * inv_dist * inv_dist;

        double F = G * dmass[i] * dmass[j] * inv_dist3;

        my_fx += F * delta_x;
        my_fy += F * delta_y;
        my_fz += F * delta_z;
    }

    dfx[i] = my_fx;
    dfy[i] = my_fy;
    dfz[i] = my_fz;
}


void dump_state(simulation& s) {
  s.device_to_host();
  for (size_t i = 0; i < s.nbpart; ++i) {
      std::cout << "Particle " << i << ":\n";
      std::cout << "  Mass: " << s.hmass[i] << " kg\n";
      std::cout << "  Position: (" << s.hx[i] << ", " << s.hy[i] << ", " << s.hz[i] << ") m\n";
      std::cout << "  Velocity: (" << s.hvx[i] << ", " << s.hvy[i] << ", " << s.hvz[i] << ") m/s\n";
      std::cout << "  Force: (" << s.hfx[i] << ", " << s.hfy[i] << ", " << s.hfz[i] << ") N\n";
      std::cout << std::endl;
  }
}

void load_from_file(simulation& s, std::string filename) {
  std::ifstream in (filename);
  size_t nbpart;
  in>>nbpart;
  s = simulation(nbpart);
  for (size_t i=0; i<s.nbpart; ++i) {
    in>>s.hmass[i];
    in >> s.hx[i] >>  s.hy[i] >>  s.hz[i];
    in >> s.hvx[i] >> s.hvy[i] >> s.hvz[i];
    in >> s.hfx[i] >> s.hfy[i] >> s.hfz[i];
  }

  s.host_to_device();
}

int main(int argc, char* argv[]) {
  if (argc != 6) {
    std::cerr
      <<"usage: "<<argv[0]<<" <input> <dt> <nbstep> <printevery>"<<"\n"
      <<"input can be:"<<"\n"
      <<"a number (random initialization)"<<"\n"
      <<"planet (initialize with solar system)"<<"\n"
      <<"a filename (load from file in singleline tsv)"<<"\n"
      <<"block size"<<"\n";
    return -1;
  }

  double dt = std::atof(argv[2]); //in seconds
  size_t nbstep = std::atol(argv[3]);
  size_t printevery = std::atol(argv[4]);
  int blockSize = std::atol(argv[5]);
  
  simulation s(1);

  //parse command line
  {
    size_t nbpart = std::atol(argv[1]); //return 0 if not a number
    if ( nbpart > 0) {
      s = simulation(nbpart);
      random_init(s);
    } else {
      std::string inputparam = argv[1];
      if (inputparam == "planet") {
	init_solar(s);
      } else{
	load_from_file(s, inputparam);
      }
    }    
  }

  
  int numBlocks = (s.nbpart + blockSize - 1) / blockSize;

  auto start = std::chrono::high_resolution_clock::now();
  
  for (size_t step = 0; step < nbstep; step++) {
      if (step % printevery == 0) {
          dump_state(s);
      }

      reset_forces_kernel<<<numBlocks, blockSize>>>(s.dfx, s.dfy, s.dfz, s.nbpart);
      
      // Compute forces on device
      compute_forces_kernel<<<numBlocks, blockSize>>>(s.dmass, s.dx, s.dy, s.dz, 
                                                    s.dfx, s.dfy, s.dfz, 
                                                    s.nbpart, G);
      
      // Update positions and velocities
      update_particles_kernel<<<numBlocks, blockSize>>>(s.dx, s.dy, s.dz,
                                                       s.dvx, s.dvy, s.dvz,
                                                       s.dfx, s.dfy, s.dfz,
                                                       s.dmass, s.nbpart, dt);
      
  }
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
  }
  hipDeviceSynchronize();

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::cout << "GPU Time: " << elapsed.count() << " s" << std::endl;
}
