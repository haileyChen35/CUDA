#include <iostream>
#include <fstream>
#include <random>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>
#include <string>
#include <cstdlib>


double G = 6.674*std::pow(10,-11);
//double G = 1;

struct simulation {
  size_t nbpart;
  
  //host 
  double* hmass;
  //position
  double* hx;
  double* hy;
  double* hz;

  //velocity
  double* hvx;
  double* hvy;
  double* hvz;

  //force
  double* hfx;
  double* hfy;
  double* hfz;

  // device
  double* dmass;
      //position
  double* dx;
  double* dy;
  double* dz;

  //velocity
  double* dvx;
  double* dvy;
  double* dvz;

  //force
  double* dfx;
  double* dfy;
  double* dfz;
  

  
  simulation(size_t nb) : nbpart(nb) {
    
    //allocate host memory
    hmass = new double[nb]();
    hx = new double[nb]();
    hy = new double[nb](); 
    hz = new double[nb]();
    hvx = new double[nb](); 
    hvy = new double[nb](); 
    hvz = new double[nb]();
    hfx = new double[nb](); 
    hfy = new double[nb](); 
    hfz = new double[nb]();

    //allocate device memory
    hipMalloc(&dmass, nb*sizeof(double));
    hipMalloc(&dx, nb * sizeof(double));
    hipMalloc(&dy, nb * sizeof(double));
    hipMalloc(&dz, nb * sizeof(double));
    hipMalloc(&dvx, nb * sizeof(double));
    hipMalloc(&dvy, nb * sizeof(double));
    hipMalloc(&dvz, nb * sizeof(double));
    hipMalloc(&dfx, nb * sizeof(double));
    hipMalloc(&dfy, nb * sizeof(double));
    hipMalloc(&dfz, nb * sizeof(double));

    //initialize memory 
    hipMemset(dmass, 0, nb * sizeof(double));
    hipMemset(dx, 0, nb * sizeof(double));
    hipMemset(dy, 0, nb * sizeof(double));
    hipMemset(dz, 0, nb * sizeof(double));
    hipMemset(dvx, 0, nb * sizeof(double));
    hipMemset(dvy, 0, nb * sizeof(double));
    hipMemset(dvz, 0, nb * sizeof(double));
    hipMemset(dfx, 0, nb * sizeof(double));
    hipMemset(dfy, 0, nb * sizeof(double));

  }

    //release memory
    ~simulation(){
        delete[] hmass;
        delete[] hx; 
        delete[] hy; 
        delete[] hz;
        delete[] hvx; 
        delete[] hvy; 
        delete[] hvz;
        delete[] hfx; 
        delete[] hfy; 
        delete[] hfz;

        hipFree(dmass);
        hipFree(dx);
        hipFree(dy);
        hipFree(dz);
        hipFree(dvx);
        hipFree(dvy);
        hipFree(dvz);
        hipFree(dfx);
        hipFree(dfy);
        hipFree(dfz);

    }

    void resize(size_t new_nbpart) {
        if (new_nbpart == nbpart) return;
    
        this->~simulation();
    
        nbpart = new_nbpart;
        hmass = new double[nbpart]();
        hx = new double[nbpart](); 
        hy = new double[nbpart](); 
        hz = new double[nbpart]();
        hvx = new double[nbpart](); 
        hvy = new double[nbpart](); 
        hvz = new double[nbpart]();
        hfx = new double[nbpart](); 
        hfy = new double[nbpart](); 
        hfz = new double[nbpart]();
    
        hipMalloc(&dmass, nbpart * sizeof(double));
        hipMalloc(&dx, nbpart * sizeof(double));
        hipMalloc(&dy, nbpart * sizeof(double));
        hipMalloc(&dz, nbpart * sizeof(double));
        hipMalloc(&dvx, nbpart * sizeof(double));
        hipMalloc(&dvy, nbpart * sizeof(double));
        hipMalloc(&dvz, nbpart * sizeof(double));
        hipMalloc(&dfx, nbpart * sizeof(double));
        hipMalloc(&dfy, nbpart * sizeof(double));
        hipMalloc(&dfz, nbpart * sizeof(double));
    
        hipMemset(dmass, 0, nbpart * sizeof(double));
        hipMemset(dx, 0, nbpart * sizeof(double));
        hipMemset(dy, 0, nbpart * sizeof(double));
        hipMemset(dz, 0, nbpart * sizeof(double));
        hipMemset(dvx, 0, nbpart * sizeof(double));
        hipMemset(dvy, 0, nbpart * sizeof(double));
        hipMemset(dvz, 0, nbpart * sizeof(double));
        hipMemset(dfx, 0, nbpart * sizeof(double));
        hipMemset(dfy, 0, nbpart * sizeof(double));
        hipMemset(dfz, 0, nbpart * sizeof(double));
      }
    
    void host_to_device() {
        hipMemcpy(dmass, hmass, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dx, hx, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dy, hy, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dz, hz, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dvx, hvx, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dvy, hvy, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dvz, hvz, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dfx, hfx, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dfy, hfy, nbpart * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dfz, hfz, nbpart * sizeof(double), hipMemcpyHostToDevice);
    }

    void device_to_host() {
      hipMemcpy(hmass, dmass, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hx, dx, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hy, dy, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hz, dz, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hvx, dvx, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hvy, dvy, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hvz, dvz, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hfx, dfx, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hfy, dfy, nbpart * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hfz, dfz, nbpart * sizeof(double), hipMemcpyDeviceToHost);
  }

   
};



void random_init(simulation& s) {
  std::random_device rd;  
  std::mt19937 gen(rd());
  std::uniform_real_distribution<double> dismass(0.9, 1.0);
  std::normal_distribution<double> dispos(0.0, 1.0);
  std::normal_distribution<double> disvel(0.0, 1.0);

  for (size_t i = 0; i<s.nbpart; ++i) {
    s.hmass[i] = dismass(gen);

    s.hx[i] = dispos(gen);
    s.hy[i] = dispos(gen);
    s.hz[i] = dispos(gen);
    s.hz[i] = 0.;
    
    s.hvx[i] = disvel(gen);
    s.hvy[i] = disvel(gen);
    s.hvz[i] = disvel(gen);
    s.hvz[i] = 0.;
    s.hvx[i] = s.hy[i]*1.5;
    s.hvy[i] = -s.hx[i]*1.5;
    s.hfx[i] = s.hfy[i] = s.hfz[i] = 0.0;

  }
    s.host_to_device();
}

void init_solar(simulation& s) {
  enum Planets {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE, MOON};
  s = simulation(10);

  // Masses in kg
  s.hmass[SUN] = 1.9891 * std::pow(10, 30);
  s.hmass[MERCURY] = 3.285 * std::pow(10, 23);
  s.hmass[VENUS] = 4.867 * std::pow(10, 24);
  s.hmass[EARTH] = 5.972 * std::pow(10, 24);
  s.hmass[MARS] = 6.39 * std::pow(10, 23);
  s.hmass[JUPITER] = 1.898 * std::pow(10, 27);
  s.hmass[SATURN] = 5.683 * std::pow(10, 26);
  s.hmass[URANUS] = 8.681 * std::pow(10, 25);
  s.hmass[NEPTUNE] = 1.024 * std::pow(10, 26);
  s.hmass[MOON] = 7.342 * std::pow(10, 22);

  // Positions (in meters) and velocities (in m/s)
  double AU = 1.496 * std::pow(10, 11); // Astronomical Unit

  s.hx[SUN] = 0; s.hy[SUN] = 0; s.hz[SUN] = 0;
  s.hx[MERCURY] = 0.39*AU; s.hy[MERCURY] = 0; s.hz[MERCURY] = 0;
  s.hx[VENUS] = 0.72*AU; s.hy[VENUS] = 0; s.hz[VENUS] = 0;
  s.hx[EARTH] = 1.0*AU; s.hy[EARTH] = 0; s.hz[EARTH] = 0;
  s.hx[MARS] = 1.52*AU; s.hy[MARS] = 0; s.hz[MARS] = 0;
  s.hx[JUPITER] = 5.20*AU; s.hy[JUPITER] = 0; s.hz[JUPITER] = 0;
  s.hx[SATURN] = 9.58*AU; s.hy[SATURN] = 0; s.hz[SATURN] = 0;
  s.hx[URANUS] = 19.22*AU; s.hy[URANUS] = 0; s.hz[URANUS] = 0;
  s.hx[NEPTUNE] = 30.05*AU; s.hy[NEPTUNE] = 0; s.hz[NEPTUNE] = 0;
  s.hx[MOON] = 1.0*AU + 3.844*std::pow(10, 8); s.hy[MOON] = 0; s.hz[MOON] = 0;

  s.hvx[SUN] = 0; s.hvy[SUN] = 0; s.hvz[SUN] = 0;
  s.hvx[MERCURY] = 0; s.hvy[MERCURY] = 47870; s.hvz[MERCURY] = 0;
  s.hvx[VENUS] = 0; s.hvy[VENUS] = 35020; s.hvz[VENUS] = 0;
  s.hvx[EARTH] = 0; s.hvy[EARTH] = 29780; s.hvz[EARTH] = 0;
  s.hvx[MARS] = 0; s.hvy[MARS] = 24130; s.hvz[MARS] = 0;
  s.hvx[JUPITER] = 0; s.hvy[JUPITER] = 13070; s.hvz[JUPITER] = 0;
  s.hvx[SATURN] = 0; s.hvy[SATURN] = 9680; s.hvz[SATURN] = 0;
  s.hvx[URANUS] = 0; s.hvy[URANUS] = 6800; s.hvz[URANUS] = 0;
  s.hvx[NEPTUNE] = 0; s.hvy[NEPTUNE] = 5430; s.hvz[NEPTUNE] = 0;
  s.hvx[MOON] = 0; s.hvy[MOON] = 29780 + 1022; s.hvz[MOON] = 0;

  for (int i = 0; i < 10; i++) {
    s.hfx[i] = 0;
    s.hfy[i] = 0;
    s.hfz[i] = 0;
    }   

    s.host_to_device();
}

//cuda kernel 
__global__ void update_particles_kernel(double* dx, double* dy, double* dz,
    double* dvx, double* dvy, double* dvz,
    double* dfx, double* dfy, double* dfz,
    double* dmass, size_t nbpart, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nbpart) return;

    dvx[i] += dfx[i] / dmass[i] * dt;
    dvy[i] += dfy[i] / dmass[i] * dt;
    dvz[i] += dfz[i] / dmass[i] * dt;

    dx[i] += dvx[i] * dt;
    dy[i] += dvy[i] * dt;
    dz[i] += dvz[i] * dt;
}

__global__ void reset_forces_kernel(double* dfx, double* dfy, double* dfz, size_t nbpart) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  dfx[i] = 0.;
  dfy[i] = 0.;
  dfz[i] = 0.;
}

__global__ void compute_forces_kernel(double* dmass, double* dx, double* dy, double* dz, 
    double* dfx, double* dfy, double* dfz, 
    size_t nbpart, double G) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nbpart) return;

    double softening = 0.1;
    double my_x = dx[i];
    double my_y = dy[i];
    double my_z = dz[i];
    double my_fx = 0.0;
    double my_fy = 0.0;
    double my_fz = 0.0;

    for (int j = 0; j < nbpart; j++) {
        if (i == j) continue;

        double delta_x = dx[j] - my_x;
        double delta_y = dy[j] - my_y;
        double delta_z = dz[j] - my_z;

        double dist_sq = delta_x*delta_x + delta_y*delta_y + delta_z*delta_z + softening;
        double inv_dist = rsqrt(dist_sq);
        double inv_dist3 = inv_dist * inv_dist * inv_dist;

        double F = G * dmass[i] * dmass[j] * inv_dist3;

        my_fx += F * delta_x;
        my_fy += F * delta_y;
        my_fz += F * delta_z;
    }

    dfx[i] = my_fx;
    dfy[i] = my_fy;
    dfz[i] = my_fz;
}


void dump_state(simulation& s) {
  std::cout<<s.nbpart<<'\t';
  for (size_t i=0; i<s.nbpart; ++i) {
    std::cout<<s.hmass[i]<<'\t';
    std::cout<<s.hx[i]<<'\t'<<s.hy[i]<<'\t'<<s.hz[i]<<'\t';
    std::cout<<s.hvx[i]<<'\t'<<s.hvy[i]<<'\t'<<s.hvz[i]<<'\t';
    std::cout<<s.hfx[i]<<'\t'<<s.hfy[i]<<'\t'<<s.hfz[i]<<'\t';
  }
  std::cout<<'\n';
}

void load_from_file(simulation& s, std::string filename) {
  std::ifstream in (filename);
  size_t nbpart;
  in>>nbpart;
  s = simulation(nbpart);
  for (size_t i=0; i<s.nbpart; ++i) {
    in>>s.hmass[i];
    in >> s.hx[i] >>  s.hy[i] >>  s.hz[i];
    in >> s.hvx[i] >> s.hvy[i] >> s.hvz[i];
    in >> s.hfx[i] >> s.hfy[i] >> s.hfz[i];
  }
  if (!in.good())
    throw "kaboom";
}

int main(int argc, char* argv[]) {
  if (argc != 6) {
    std::cerr
      <<"usage: "<<argv[0]<<" <input> <dt> <nbstep> <printevery>"<<"\n"
      <<"input can be:"<<"\n"
      <<"a number (random initialization)"<<"\n"
      <<"planet (initialize with solar system)"<<"\n"
      <<"a filename (load from file in singleline tsv)"<<"\n"
      <<"block size"<<"\n";
    return -1;
  }

  double dt = std::atof(argv[2]); //in seconds
  size_t nbstep = std::atol(argv[3]);
  size_t printevery = std::atol(argv[4]);
  int blockSize = std::atol(argv[5]);
  
  simulation s(1);

  //parse command line
  {
    size_t nbpart = std::atol(argv[1]); //return 0 if not a number
    if ( nbpart > 0) {
      s = simulation(nbpart);
      random_init(s);
    } else {
      std::string inputparam = argv[1];
      if (inputparam == "planet") {
	init_solar(s);
      } else{
	load_from_file(s, inputparam);
      }
    }    
  }

  
  int numBlocks = (s.nbpart + blockSize - 1) / blockSize;

  auto start = std::chrono::high_resolution_clock::now();
  
  for (size_t step = 0; step < nbstep; step++) {
      if (step % printevery == 0) {
          s.device_to_host();
          dump_state(s);
      }

      reset_forces_kernel<<<numBlocks, blockSize>>>(s.dfx, s.dfy, s.dfz, s.nbpart);
      
      // Compute forces on device
      compute_forces_kernel<<<numBlocks, blockSize>>>(s.dmass, s.dx, s.dy, s.dz, 
                                                    s.dfx, s.dfy, s.dfz, 
                                                    s.nbpart, G);
      
      // Update positions and velocities
      update_particles_kernel<<<numBlocks, blockSize>>>(s.dx, s.dy, s.dz,
                                                       s.dvx, s.dvy, s.dvz,
                                                       s.dfx, s.dfy, s.dfz,
                                                       s.dmass, s.nbpart, dt);
      
  }
  
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::cout << "GPU Time: " << elapsed.count() << " s" << std::endl;
}
